#include "hip/hip_runtime.h"
#include <cassert>
#include <random>
#include <lapacke.h>
#include <cutf/memory.hpp>
#include <cutf/cusolver.hpp>
#include <mateval/comparison_cuda.hpp>

struct svdj_params {
	float ctol;
};

void lapack_svdj(
		const std::size_t m,
		const std::size_t n,
		const float* const host_A, // M x N
		float* const host_U, // M x N
		float* const host_S, // N
		float* const host_V, // N x N
		const svdj_params params
		) {
#pragma omp parallel for
	for (std::size_t i = 0; i < m * n; i++) {
		host_U[i] = host_A[i];
	}

	const auto lwork = std::max<std::size_t>(6, m + n);
	auto work_uptr = cutf::memory::get_host_unique_ptr<float>(lwork);

	//lapack_int LAPACKE_sgesvj_work( int matrix_layout, char joba, char jobu,
	//                                char jobv, lapack_int m, lapack_int n, float* a,
	//                                lapack_int lda, float* sva, lapack_int mv,
	//                                float* v, lapack_int ldv, float* work,
	//                                lapack_int lwork );
	//lapack_int LAPACKE_sgesvj( int matrix_layout, char joba, char jobu, char jobv,
	//                           lapack_int m, lapack_int n, float* a, lapack_int lda,
	//                           float* sva, lapack_int mv, float* v, lapack_int ldv,
	//                           float* stat );
	work_uptr.get()[0] = params.ctol;

	const auto info = LAPACKE_sgesvj_work(
			LAPACK_COL_MAJOR,
			'G',
			'C', // 'C'?
			'V', // 'A'?
			m, n,
			host_U,
			m,
			host_S,
			n,
			host_V,
			n,
			work_uptr.get(),
			lwork);
}

void cusolver_svdj(
		const std::size_t m,
		const std::size_t n,
		const float* const host_A, // M x N
		float* const host_U, // M x N
		float* const host_S, // N
		float* const host_V, // N x N
		const svdj_params params
		) {
	auto da_uptr = cutf::memory::get_device_unique_ptr<float>(m * n);
	auto du_uptr = cutf::memory::get_device_unique_ptr<float>(m * n);
	auto dv_uptr = cutf::memory::get_device_unique_ptr<float>(n * n);
	auto ds_uptr = cutf::memory::get_device_unique_ptr<float>(n);

	cutf::memory::copy(da_uptr.get(), host_A, m * n);

	hipsolverGesvdjInfo_t svdj_params;
	const double tol = params.ctol * LAPACKE_slamch('E');
	const unsigned num_svdj_iter = 1000;
	CUTF_CHECK_ERROR(hipsolverDnCreateGesvdjInfo(&svdj_params));
	CUTF_CHECK_ERROR(hipsolverDnXgesvdjSetMaxSweeps(svdj_params, num_svdj_iter));
	CUTF_CHECK_ERROR(hipsolverDnXgesvdjSetTolerance(svdj_params, tol));


	auto cusolver_handle_uptr = cutf::cusolver::dn::get_handle_unique_ptr();
	int tmp_working_memory_size;
	CUTF_CHECK_ERROR(hipsolverDnSgesvdj_bufferSize(
				*cusolver_handle_uptr.get(),
				HIPSOLVER_EIG_MODE_VECTOR,
				1,
				m, n,
				da_uptr.get(), m,
				ds_uptr.get(),
				du_uptr.get(), m,
				dv_uptr.get(), n,
				&tmp_working_memory_size,
				svdj_params
				));

	const auto working_memory_device_size = tmp_working_memory_size;
	auto working_memory_device_uptr = cutf::memory::get_device_unique_ptr<float>(working_memory_device_size);

	auto devInfo_uptr = cutf::memory::get_device_unique_ptr<int>(1);

	CUTF_CHECK_ERROR(hipsolverDnSgesvdj(
				*cusolver_handle_uptr.get(),
				HIPSOLVER_EIG_MODE_VECTOR,
				1,
				m, n,
				da_uptr.get(), m,
				ds_uptr.get(),
				du_uptr.get(), m,
				dv_uptr.get(), n,
				working_memory_device_uptr.get(),
				working_memory_device_size,
				devInfo_uptr.get(),
				svdj_params
				));

	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	cutf::memory::copy(host_U, du_uptr.get(), m * n);
	cutf::memory::copy(host_V, dv_uptr.get(), n * n);
	cutf::memory::copy(host_S, ds_uptr.get(), n);
}

void svdj_eval(
		const std::size_t m,
		const std::size_t n,
		const std::string mode
		) {
	assert(m >= n);

	auto A_uptr = cutf::memory::get_managed_unique_ptr<float>(m * n);
	auto U_uptr = cutf::memory::get_managed_unique_ptr<float>(m * n);
	auto S_uptr = cutf::memory::get_managed_unique_ptr<float>(n);
	auto V_uptr = cutf::memory::get_managed_unique_ptr<float>(n * n);

	std::normal_distribution<float> N_dist(0.f, 1.f);
	std::mt19937 mt(0);
	for (std::size_t i = 0; i < m * n; i++) {
		A_uptr.get()[i] = N_dist(mt);
	}

	svdj_params params{.ctol = 2};

	if (mode == "lapack") {
		lapack_svdj(
				m, n,
				A_uptr.get(),
				U_uptr.get(),
				S_uptr.get(),
				V_uptr.get(),
				params
				);
	} else {
		cusolver_svdj(
				m, n,
				A_uptr.get(),
				U_uptr.get(),
				S_uptr.get(),
				V_uptr.get(),
				params
				);
	}

	const auto residual = mtk::mateval::cuda::residual_UxSxVt(
			m, n, n,
			mtk::mateval::col_major,
			mtk::mateval::col_major,
			mtk::mateval::col_major,
			U_uptr.get(), m,
			S_uptr.get(),
			V_uptr.get(), n,
			A_uptr.get(), m
			);

	const auto orth_U = mtk::mateval::cuda::orthogonality(
			m, n,
			mtk::mateval::col_major,
			U_uptr.get(), m
			);

	const auto orth_V = mtk::mateval::cuda::orthogonality(
			n, n,
			mtk::mateval::col_major,
			V_uptr.get(), n
			);

	std::printf("%s,%lu,%lu,%e,%e,%e\n",
			mode.c_str(),
			m, n,
			residual,
			orth_U,
			orth_V
			);
}

int main() {
	std::printf("mode,m,n,residual,u_orth,v_orth\n");
	for (unsigned N = 256; N <= (1u << 12); N <<= 1) {
		svdj_eval(N, N, "lapack");
		svdj_eval(N, N, "cusolver");
	}
}
